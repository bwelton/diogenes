#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <unistd.h>
#include <time.h>
#include <sys/time.h>

#define ALLOC_SIZE 1024000
#define hipSuccess hipSuccess 

int GetCudaFrequency() {
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  return prop.clockRate;
}

__global__ void SpinForever() {
  int i = 0;
  while (i == 0) {}
}

// Spin in the GPU for a certain delay, siAAmulates load to induce synchronization delay.
__global__ void LaunchSpinForTime(uint64_t delay) {
    uint64_t start_clock = clock64();
    uint64_t clock_offset = 0;
    while (clock_offset < delay)
    {
        clock_offset = clock64() - start_clock;
        //printf("Clock Offset: %llu\n", clock_offset);
    }
}


int ExperimentNoUse(uint64_t cpuDelay) {
  // 1.18 is a constant factor to delay to the exact amount of CPU Delay
  uint64_t gpuDelay = ((uint64_t(cpuDelay) * uint64_t(GetCudaFrequency() * 1000)) * 1.18) * 1.20;
  fprintf(stderr, "GPU will delay for - %llu clock cycles (%llu seconds)\n",gpuDelay, uint64_t(cpuDelay * 1.20));
  //uint64_t *x, *d_x, *d_y;
  // Launch GPU Kernel
  LaunchSpinForTime<<<1,1>>>(gpuDelay);
  //LaunchSpinOnMem<<<1,1>>>(flags);
  // Immediately sleep to simulate CPU workload
  sleep(cpuDelay);
  fprintf(stderr, "Left CPU Work, waiting on GPU synchronization\n");

  // Unnnecessary Synchronization that should take between 10-20% of execution time
  struct timeval t0,t1;
  gettimeofday(&t0, 0);
  hipDeviceSynchronize();
  gettimeofday(&t1, 0);
  long long unsigned elapsed = (t1.tv_sec-t0.tv_sec)*1000000 + t1.tv_usec-t0.tv_usec;
  fprintf(stderr, "GPU synchronization complete - Actual Delay %llu\n", elapsed/1000000);
  sleep(15);
  return 0;
}

int myexperiment() {
  uint64_t *d_x;
  if (hipSuccess != hipMalloc(&d_x, 100))
   fprintf(stderr, "ERROR\n"); 
  for (int i = 0; i < 2; i++)
    ExperimentNoUse(uint64_t(60));
  return 0;
}

int main() {
    return myexperiment();
}
