#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one or more
 * contributor license agreements.  See the NOTICE file distributed with
 * this work for additional information regarding copyright ownership.
 * The ASF licenses this file to You under the Apache License, Version 2.0
 * (the "License"); you may not use this file except in compliance with
 * the License.  You may obtain a copy of the License at
 *
 *    http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
/*
 * als.cu
 *
 *  Created on: Feb 10, 2015
 *      Author: Wei Tan (wtan@us.ibm.com)
 *  Alternating Least Square for Matrix Factorization on CUDA 7.0+
 *  Code optimized for F = 100, and on cc 3.5, 3.7 platforms. Also tested in cc 5.2
 */
//do not use fp16 by default  
//#define CUMF_USE_HALF
#define SURPASS_NAN
#define USE_CG
//if cojugate gradient solver generates results in FP16 
//#define CUMF_TT_FP16
//#define CUMF_XX_FP16
#define CG_ITER 6
//#define CUMF_SAVE_MODEL
#include "als.h"
#include "device_utilities.h"
#include "cg.h"
#include "host_utilities.h"
#include <fstream>
#include <assert.h>
#include <hip/hip_fp16.h>
#ifdef CUMF_USE_HALF
#define SCAN_BATCH 24
#else
#define SCAN_BATCH 28
#endif
#include <iostream>
using namespace std;

void saveDeviceFloatArrayToFile(string fileName, int size, float* d_array){
	float* h_array;
	cudacall(hipHostMalloc( (void** ) &h_array, size * sizeof(h_array[0])) );
	cudacall(hipMemcpy(h_array, d_array, size * sizeof(h_array[0]),hipMemcpyDeviceToHost));
	FILE * outfile = fopen(fileName.c_str(), "wb");
	fwrite(h_array, sizeof(float), size, outfile);
	fclose(outfile);
	hipHostFree(h_array);
}
int updateX(const int batch_size, const int batch_offset, float * ythetaT, float * tt, float * XT,
		hipblasHandle_t handle, const int m, const int n, const int f, const int nnz,
		float** devPtrTTHost, float **devPtrYthetaTHost){
	#ifdef DEBUG
	float elapsed;
	struct timeval tv0, tv1, tv2;
	gettimeofday(&tv0, NULL);
	printf("*******Batch LU factorization of tt.\n");
	#endif
	//pointers needed by batch op
	float **devPtrTT = 0;
	int *INFO;
	for (int k = 0; k < batch_size; k++) {
		devPtrTTHost[k] = &tt[k * f * f];
	}
	cudacall(hipMalloc((void** ) &devPtrTT, batch_size * sizeof(*devPtrTT)));
	cudacall(hipMemcpy(devPtrTT, devPtrTTHost, batch_size * sizeof(*devPtrTT),hipMemcpyHostToDevice));
	//cudacall( hipMalloc(&P, f * batch_size * sizeof(int)) );
	cudacall( hipMalloc(&INFO, batch_size * sizeof(int) ));
	cublascall(hipblasSgetrfBatched(handle, f, devPtrTT, f, NULL, INFO, batch_size));

	hipDeviceSynchronize();
	#ifdef DEBUG
	gettimeofday(&tv1, NULL);
	elapsed = (tv1.tv_sec - tv0.tv_sec)
			+ (tv1.tv_usec - tv0.tv_usec) / 1000000.0;
	printf("\t %f seconds. \n", elapsed);

	printf("*******solve: tt * XT = ythetaT use cublas, with LU decomposition.\n");
	#endif

	float **devPtrYthetaT = 0;

	for (int k = 0; k < batch_size; k++) {
		devPtrYthetaTHost[k] = &ythetaT[batch_offset * f + k * f];
	}
	cudacall(hipMalloc((void** ) &devPtrYthetaT, batch_size * sizeof(*devPtrYthetaT)));
	cudacall(hipMemcpy(devPtrYthetaT, devPtrYthetaTHost, batch_size * sizeof(*devPtrYthetaT), hipMemcpyHostToDevice));

	int * info2 = (int *) malloc(sizeof(int));
	cublascall( hipblasSgetrsBatched(handle, HIPBLAS_OP_N, f, 1,
			(const float ** ) devPtrTT, f, NULL, devPtrYthetaT, f, info2, batch_size) );

	hipDeviceSynchronize();
	hipError_t cudaStat1 = hipGetLastError();
	if (cudaStat1 != hipSuccess) {
		fprintf(stderr,"Failed to launch hipblasSgetrsBatched (error code: %s)!\n", hipGetErrorString(cudaStat1));
		exit(EXIT_FAILURE);
	}

	cudacall( hipMemcpy(&XT[batch_offset * f], &ythetaT[batch_offset * f],
			batch_size * f * sizeof(float), hipMemcpyDeviceToDevice) );
	#ifdef DEBUG
	gettimeofday(&tv2, NULL);
	elapsed = (tv2.tv_sec - tv1.tv_sec)
			+ (tv2.tv_usec - tv1.tv_usec) / 1000000.0;
	printf("\t %f seconds. \n", elapsed);
	#endif

	cudacall(hipFree(devPtrTT));
	//cudacall(hipFree(P));
	cudacall(hipFree(INFO));
	cudacall(hipFree(devPtrYthetaT));
	return 0;
}

int updateTheta(const int batch_size, const int batch_offset, float * xx,
		  float * yTXT, float * thetaT,
		hipblasHandle_t handle,
		 const int m, const int n, const int f, const int nnz,
		 float ** devPtrXXHost, float **devPtrYTXTHost ){

	#ifdef DEBUG
	float elapsed;
	struct timeval tv0, tv1, tv2;
	gettimeofday(&tv0, NULL);
	printf("*******LU factorize xx.\n");
	#endif
	float **devPtrXX = 0;

	for (int k = 0; k < batch_size; k++) {
		devPtrXXHost[k] = &xx[k * f * f];
	}
	cudacall(hipMalloc((void** ) &devPtrXX, batch_size * sizeof(*devPtrXX)));
	cudacall(hipMemcpy(devPtrXX, devPtrXXHost, batch_size * sizeof(*devPtrXX), hipMemcpyHostToDevice));
	int *INFO;
	//cudacall(hipMalloc(&P, f * batch_size * sizeof(int)));
	cudacall(hipMalloc(&INFO, batch_size * sizeof(int)));
	cublascall(hipblasSgetrfBatched(handle, f, devPtrXX, f, NULL, INFO, batch_size));
	hipDeviceSynchronize();
	#ifdef DEBUG
	gettimeofday(&tv1, NULL);
	elapsed = (tv1.tv_sec - tv0.tv_sec)
			+ (tv1.tv_usec - tv0.tv_usec) / 1000000.0;
	printf("\t %f seconds. \n", elapsed);

	printf("******* solve xx * thetaT = yTXT with CUDA 7.\n");
	#endif
	float **devPtrYTXT = 0;

	for (int k = 0; k < batch_size; k++) {
		devPtrYTXTHost[k] = &yTXT[batch_offset * f + k * f];
	}

	cudacall(hipMalloc((void** ) &devPtrYTXT, batch_size * sizeof(*devPtrYTXT)));
	cudacall(hipMemcpy(devPtrYTXT, devPtrYTXTHost, batch_size * sizeof(*devPtrYTXT),hipMemcpyHostToDevice));

	int * info2 = (int *) malloc(sizeof(int));
	cublascall( hipblasSgetrsBatched(handle, HIPBLAS_OP_N, f, 1,
			(const float ** ) devPtrXX, f, NULL, devPtrYTXT, f, info2, batch_size) );
	hipDeviceSynchronize();
	hipError_t cudaStat1 = hipGetLastError();
	if (cudaStat1 != hipSuccess) {
		fprintf(stderr,"Failed to launch hipblasSgetrsBatched (error code: %s)!\n", hipGetErrorString(cudaStat1));
		exit(EXIT_FAILURE);
	}

	cudacall( hipMemcpy( &thetaT[batch_offset * f], &yTXT[batch_offset * f],
	                        batch_size * f * sizeof(float), hipMemcpyDeviceToDevice) );
	#ifdef DEBUG
	gettimeofday(&tv2, NULL);
	elapsed = (tv2.tv_sec - tv1.tv_sec)
			+ (tv2.tv_usec - tv1.tv_usec) / 1000000.0;
	printf("\t %f seconds. \n", elapsed);
	#endif

	hipFree(devPtrXX);
	hipFree(INFO);
	free(info2);
	hipFree(devPtrYTXT);
	return 0;
}

__global__ void RMSE(const float * csrVal, const int* cooRowIndex,
		const int* csrColIndex, const float * __restrict__ thetaT, const float * __restrict__ XT, float * error, const int nnz,
		const int error_size, const int f) {
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < nnz) {
		int row = cooRowIndex[i];
		int col = csrColIndex[i];
		float e = csrVal[i];
		//if(i%1000000==0) printf("row: %d, col: %d, csrVal[%d]: %f.\n", row, col, i, e);
		for (int k = 0; k < f; k++) {
			#ifdef SURPASS_NAN
			//a and b could be; there are user/item in testing but not training set
			float a = __ldg(&thetaT[f * col + k]);
			float b = __ldg(&XT[f * row + k]);
			//if(isnan(a)||isnan(b))//nan not working in some platform
			if(a!=a||b!=b)
				break;
			else
				e -= a * b;
			//if(isnan(a)) printf("row: %d, col: %d\n", row, col);
			//if(isnan(b)) printf("b[%d]: %f.\n", i, b);
			#else
			e -= __ldg(&thetaT[f * col + k]) * __ldg(&XT[f * row + k]);
			#endif
		}
		atomicAdd(&error[i%error_size], e*e);
		//if(i%1000000==0) printf("error[%d]: %f.\n", i, e);
	}
}

//using fp16 as thetaT's format
//using fp16 in computate seems causing register pressure since half intrinsics cannot be used.
//using fp16 in compute also does not converge. not sure if the code is incorrect, or ALS cannot tolerate half-precision
__global__ void
__launch_bounds__(64, 6)
get_hermitian100WithHalf(const int batch_offset, float* tt,
		const int* csrRowIndex, const int* csrColIndex, const float lambda, const int m, const int F,
		const half* __restrict__ thetaT_fp16) {
	extern __shared__ float2 thetaTemp[];
	int row = blockIdx.x + batch_offset;
	if (row < m) {
		//this block needs to handle end - start thetaT columns
		int start = csrRowIndex[row];
		int end = csrRowIndex[row + 1];
		//slide through [start, end] by window size SCAN_BATCH
		int iterations = (end - start - 1)/SCAN_BATCH + 1;
		
		float temp0= 0, temp1= 0, temp2= 0, temp3= 0, temp4= 0, temp5= 0, temp6= 0, temp7= 0, temp8= 0, temp9 = 0;
		float temp10= 0, temp11= 0, temp12= 0, temp13= 0, temp14= 0, temp15= 0, temp16= 0, temp17= 0, temp18= 0, temp19 = 0;
		float temp20= 0, temp21= 0, temp22= 0, temp23= 0, temp24= 0, temp25= 0, temp26= 0, temp27= 0, temp28= 0, temp29 = 0;
		float temp30= 0, temp31= 0, temp32= 0, temp33= 0, temp34= 0, temp35= 0, temp36= 0, temp37= 0, temp38= 0, temp39 = 0;
		float temp40= 0, temp41= 0, temp42= 0, temp43= 0, temp44= 0, temp45= 0, temp46= 0, temp47= 0, temp48= 0, temp49 = 0;
		float temp50= 0, temp51= 0, temp52= 0, temp53= 0, temp54= 0, temp55= 0, temp56= 0, temp57= 0, temp58= 0, temp59 = 0;
		float temp60= 0, temp61= 0, temp62= 0, temp63= 0, temp64= 0, temp65= 0, temp66= 0, temp67= 0, temp68= 0, temp69 = 0;
		float temp70= 0, temp71= 0, temp72= 0, temp73= 0, temp74= 0, temp75= 0, temp76= 0, temp77= 0, temp78= 0, temp79 = 0;
		float temp80= 0, temp81= 0, temp82= 0, temp83= 0, temp84= 0, temp85= 0, temp86= 0, temp87= 0, temp88= 0, temp89 = 0;
		float temp90= 0, temp91= 0, temp92= 0, temp93= 0, temp94= 0, temp95= 0, temp96= 0, temp97= 0, temp98= 0, temp99 = 0;
	
		int tile_x = 0;
		int tile_y = 0;

		int tile = F/10;
		for ( int i = 0; i < 10; i++){
			int end = ((20-i)*(i+1))/2;
			if(threadIdx.x < end){
				tile_x = i * tile;
				tile_y = (10 + threadIdx.x - end) * tile;
				break;
			}
		}
		//iteration: copy gmem-->smem; aggregate smem-->register
		for (int iter = 0; iter < iterations; iter ++){
			//float2 theta;
			//copy texture --> smem, and sync
			//two layers: warp divergence unless we split at 32
			//require: 32 >= SCAN_BATCH
			if(threadIdx.x < 2*32 ){
				int index = threadIdx.x - (threadIdx.x/32)*32;	//0 to 31;
				if(index < SCAN_BATCH){
					if(iter*SCAN_BATCH + index < end - start){
						//for (int k = 50*(threadIdx.x/32); k < 50*(threadIdx.x/32) + 50; k += 2){
						//IMPORTANT: for loop has constant and identical start and end
						if(threadIdx.x < 32){
							for (int k = 0; k < 50; k += 2){
								half2 theta_half2 = __ldg((half2*)&thetaT_fp16[ F * csrColIndex[start + iter*SCAN_BATCH + index] + k]);
								thetaTemp[index * F/2 + k/2] = __half22float2(theta_half2);
								//theta.x = __half2float(__ldg(&thetaT_fp16[ F * csrColIndex[start + iter*SCAN_BATCH + index] + k]));
								//theta.y = __half2float(__ldg(&thetaT_fp16[ F * csrColIndex[start + iter*SCAN_BATCH + index] + k+1]));
								//thetaTemp[index * F/2 + k/2] = theta;
							}
						}
						else {
							for (int k = 0; k < 50; k += 2){
								half2 theta_half2 = __ldg((half2*)&thetaT_fp16[ F * csrColIndex[start + iter*SCAN_BATCH + index] + k + 50]);
								thetaTemp[index * F/2 + k/2 + 25] = __half22float2(theta_half2);
								//theta.x = __half2float(__ldg(&thetaT_fp16[ F * csrColIndex[start + iter*SCAN_BATCH + index] + k + 50]));
								//theta.y = __half2float(__ldg(&thetaT_fp16[ F * csrColIndex[start + iter*SCAN_BATCH + index] + k + 51]));
								//thetaTemp[index * F/2 + k/2 + 25] = theta;
							}
						}
					}
					//must be the last iteration; no need to check
					//not enough theta to copy, set zero
					else
						memset(&thetaTemp[index*F/2], 0, F*sizeof(float));
				}
			}
			__syncthreads();
			//tile: 10*10
			if(threadIdx.x < 55 ){
				for(int k = 0; k < SCAN_BATCH; k++){
					accumulate_in_registers();
				}
			}
		}
		//end of iteration in copying from smem and aggregating in register
		__syncthreads();

		if(threadIdx.x < 55 ){
			//weighted-lambda regularization
			if(tile_x == tile_y){
				float temp = (end - start) * lambda;
				temp0 += temp;
				temp11 += temp;
				temp22 += temp;
				temp33 += temp;
				temp44 += temp;
				temp55 += temp;
				temp66 += temp;
				temp77 += temp;
				temp88 += temp;
				temp99 += temp;
			}
			//copy output to gmem
			int index = blockIdx.x*F*F;
			fill_lower_half_from_registers();
			//symmetric
			if(tile_x!=tile_y){
				fill_upper_half_from_registers();
			}
		}
	}
}

__global__ void
__launch_bounds__(64, 6)
get_hermitian100_tt_fp16(const int batch_offset, half2* tt,
		const int* csrRowIndex, const int* csrColIndex, const float lambda, const int m, const int F,
		const float2* __restrict__ thetaT) {
	extern __shared__ float2 thetaTemp[];
	int row = blockIdx.x + batch_offset;
	if (row < m) {
		//this block needs to handle end - start thetaT columns
		int start = csrRowIndex[row];
		int end = csrRowIndex[row + 1];
		//slide through [start, end] by window size SCAN_BATCH
		int iterations = (end - start - 1)/SCAN_BATCH + 1;
		float temp0= 0, temp1= 0, temp2= 0, temp3= 0, temp4= 0, temp5= 0, temp6= 0, temp7= 0, temp8= 0, temp9 = 0;
		float temp10= 0, temp11= 0, temp12= 0, temp13= 0, temp14= 0, temp15= 0, temp16= 0, temp17= 0, temp18= 0, temp19 = 0;
		float temp20= 0, temp21= 0, temp22= 0, temp23= 0, temp24= 0, temp25= 0, temp26= 0, temp27= 0, temp28= 0, temp29 = 0;
		float temp30= 0, temp31= 0, temp32= 0, temp33= 0, temp34= 0, temp35= 0, temp36= 0, temp37= 0, temp38= 0, temp39 = 0;
		float temp40= 0, temp41= 0, temp42= 0, temp43= 0, temp44= 0, temp45= 0, temp46= 0, temp47= 0, temp48= 0, temp49 = 0;
		float temp50= 0, temp51= 0, temp52= 0, temp53= 0, temp54= 0, temp55= 0, temp56= 0, temp57= 0, temp58= 0, temp59 = 0;
		float temp60= 0, temp61= 0, temp62= 0, temp63= 0, temp64= 0, temp65= 0, temp66= 0, temp67= 0, temp68= 0, temp69 = 0;
		float temp70= 0, temp71= 0, temp72= 0, temp73= 0, temp74= 0, temp75= 0, temp76= 0, temp77= 0, temp78= 0, temp79 = 0;
		float temp80= 0, temp81= 0, temp82= 0, temp83= 0, temp84= 0, temp85= 0, temp86= 0, temp87= 0, temp88= 0, temp89 = 0;
		float temp90= 0, temp91= 0, temp92= 0, temp93= 0, temp94= 0, temp95= 0, temp96= 0, temp97= 0, temp98= 0, temp99 = 0;

		int tile_x = 0;
		int tile_y = 0;

		int tile = F/10;
		for ( int i = 0; i < 10; i++){
			int end = ((20-i)*(i+1))/2;
			if(threadIdx.x < end){
				tile_x = i * tile;
				tile_y = (10 + threadIdx.x - end) * tile;
				break;
			}
		}
		//iteration: copy gmem-->smem; aggregate smem-->register
		for (int iter = 0; iter < iterations; iter ++){
			//copy texture --> smem, and sync
			/*
			This is the fastest implementation
			thetaT is NOT coalesced loaded but cached by L1 and L2
			faster than coalesced version (see the next paragraph commented out) 
			because it concurrently load multiple thetaT columns
			two threads per theta column, e.g., threads 0 & 1 for theta[0], threads 2 & 3 for theta[1]
			require: blockDim.x (64) >= 2*SCAN_BATCH
			*/
///* 
			if(threadIdx.x < 2*SCAN_BATCH){
				int anchor = start + iter*SCAN_BATCH + threadIdx.x/2;
				if(anchor < end){
					int col = csrColIndex[anchor];
					//IMPORTANT: for loop has constant and identical start and end
					for (int k = 0; k < 50; k += 2)
						//thetaTemp[threadIdx.x*F/4 + k/2] =__ldg(&thetaT[ F/2 * col + threadIdx.x%2*F/4 + k/2]);
						thetaTemp[threadIdx.x*F/4 + k/2] = thetaT[ F/2 * col + threadIdx.x%2*F/4 + k/2];
				}
			}
//*/
			__syncthreads();

			//tile: 10*10
			if(threadIdx.x < 55){
				if(iter < iterations - 1){
					for(int k = 0; k < SCAN_BATCH; k++)
						accumulate_in_registers();
				}
				else{
					for(int k = 0; k < end - start - iter*SCAN_BATCH; k++)
						accumulate_in_registers();
				}
				
			}
		}
		//end of iteration in copying from smem and aggregating in register
		__syncthreads();
		#ifdef DEBUG
		//if(threadIdx.x==0)
		//	printf("***temp 0~9: %f %f %f %f %f %f %f %f %f %f\n", temp0, temp1, temp2, temp3, temp4, temp5, temp6, temp7, temp8, temp9);
		#endif
		if(threadIdx.x < 55 ){
			//weighted-lambda regularization
			if(tile_x == tile_y){
				float temp = (end - start) * lambda;
				temp0 += temp;
				temp11 += temp;
				temp22 += temp;
				temp33 += temp;
				temp44 += temp;
				temp55 += temp;
				temp66 += temp;
				temp77 += temp;
				temp88 += temp;
				temp99 += temp;
			}
			//copy output to gmem
			int index = blockIdx.x*F*F/2;
			//fill_lower_half_from_registers();
			fill_lower_half_from_registers_fp16();
			//symmetric
			if(tile_x!=tile_y){
				//fill_upper_half_from_registers();
				fill_upper_half_from_registers_fp16();
			}
		}
	}
}

__global__ void
__launch_bounds__(64)
get_hermitian100(const int batch_offset, float2* tt,
		const int* csrRowIndex, const int* csrColIndex, const float lambda, const int m, const int F,
		const float2* __restrict__ thetaT) {
	extern __shared__ float2 thetaTemp[];
	int row = blockIdx.x + batch_offset;
	if (row < m) {
		//this block needs to handle end - start thetaT columns
		int start = csrRowIndex[row];
		int end = csrRowIndex[row + 1];
		//slide through [start, end] by window size SCAN_BATCH
		int iterations = (end - start - 1)/SCAN_BATCH + 1;
		float temp0= 0, temp1= 0, temp2= 0, temp3= 0, temp4= 0, temp5= 0, temp6= 0, temp7= 0, temp8= 0, temp9 = 0;
		float temp10= 0, temp11= 0, temp12= 0, temp13= 0, temp14= 0, temp15= 0, temp16= 0, temp17= 0, temp18= 0, temp19 = 0;
		float temp20= 0, temp21= 0, temp22= 0, temp23= 0, temp24= 0, temp25= 0, temp26= 0, temp27= 0, temp28= 0, temp29 = 0;
		float temp30= 0, temp31= 0, temp32= 0, temp33= 0, temp34= 0, temp35= 0, temp36= 0, temp37= 0, temp38= 0, temp39 = 0;
		float temp40= 0, temp41= 0, temp42= 0, temp43= 0, temp44= 0, temp45= 0, temp46= 0, temp47= 0, temp48= 0, temp49 = 0;
		float temp50= 0, temp51= 0, temp52= 0, temp53= 0, temp54= 0, temp55= 0, temp56= 0, temp57= 0, temp58= 0, temp59 = 0;
		float temp60= 0, temp61= 0, temp62= 0, temp63= 0, temp64= 0, temp65= 0, temp66= 0, temp67= 0, temp68= 0, temp69 = 0;
		float temp70= 0, temp71= 0, temp72= 0, temp73= 0, temp74= 0, temp75= 0, temp76= 0, temp77= 0, temp78= 0, temp79 = 0;
		float temp80= 0, temp81= 0, temp82= 0, temp83= 0, temp84= 0, temp85= 0, temp86= 0, temp87= 0, temp88= 0, temp89 = 0;
		float temp90= 0, temp91= 0, temp92= 0, temp93= 0, temp94= 0, temp95= 0, temp96= 0, temp97= 0, temp98= 0, temp99 = 0;

		int tile_x = 0;
		int tile_y = 0;

		int tile = F/10;
		for ( int i = 0; i < 10; i++){
			int end = ((20-i)*(i+1))/2;
			if(threadIdx.x < end){
				tile_x = i * tile;
				tile_y = (10 + threadIdx.x - end) * tile;
				break;
			}
		}
		//iteration: copy gmem-->smem; aggregate smem-->register
		for (int iter = 0; iter < iterations; iter ++){
			//copy texture --> smem, and sync
			/*
			This is the fastest implementation
			thetaT is NOT coalesced loaded but cached by L1 and L2
			faster than coalesced version (see the next paragraph commented out) 
			because it concurrently load multiple thetaT columns
			two threads per theta column, e.g., threads 0 & 1 for theta[0], threads 2 & 3 for theta[1]
			require: blockDim.x (64) >= 2*SCAN_BATCH
			*/
///* 
			if(threadIdx.x < 2*SCAN_BATCH){
				int anchor = start + iter*SCAN_BATCH + threadIdx.x/2;
				if(anchor < end){
					int col = csrColIndex[anchor];
					//IMPORTANT: for loop has constant and identical start and end
					for (int k = 0; k < 50; k += 2)
						//thetaTemp[threadIdx.x*F/4 + k/2] =__ldg(&thetaT[ F/2 * col + threadIdx.x%2*F/4 + k/2]);
						thetaTemp[threadIdx.x*F/4 + k/2] = thetaT[ F/2 * col + threadIdx.x%2*F/4 + k/2];
				}
			}
//*/			

/*			
				//coalesced load thetaT, has to load column by column, less concurrency, worse performance
				int anchor = start + iter*SCAN_BATCH + threadIdx.x%32;
				int col_local;
				if(anchor < end && threadIdx.x%32 < SCAN_BATCH)
					col_local = csrColIndex[anchor];
				int stop = (end - start - iter*SCAN_BATCH < SCAN_BATCH)? end - start - iter*SCAN_BATCH: SCAN_BATCH;
				for (int k = 0; k < stop; k++){
					//deal with col_local in lane[k]
					int col = __shfl(col_local, k);
					//if(blockIdx.x==0 && threadIdx.x==0)
					//	printf("iter=%d,k=%d,col=%d,stop=%d,anchor=%d\n", iter,k, col, stop, anchor);
					//this type of for is bad in performance
					//for(int i = threadIdx.x; i < F; i += 64)
					if(threadIdx.x<F/2)
						thetaTemp[k*F/2 + threadIdx.x] = __ldg(&thetaT[ F/2 * col + threadIdx.x]);
				}
*/
			__syncthreads();
///*
			//tile: 10*10
			if(threadIdx.x < 55){
				if(iter < iterations - 1){
					for(int k = 0; k < SCAN_BATCH; k++)
						accumulate_in_registers();
				}
				else{
					for(int k = 0; k < end - start - iter*SCAN_BATCH; k++)
						accumulate_in_registers();
				}
				
			}
//*/			
		}
		//end of iteration in copying from smem and aggregating in register
		__syncthreads();
		#ifdef DEBUG
		//if(threadIdx.x==0)
		//	printf("***temp 0~9: %f %f %f %f %f %f %f %f %f %f\n", temp0, temp1, temp2, temp3, temp4, temp5, temp6, temp7, temp8, temp9);
		#endif
		if(threadIdx.x < 55 ){
			//weighted-lambda regularization
			if(tile_x == tile_y){
				float temp = (end - start) * lambda;
				temp0 += temp;
				temp11 += temp;
				temp22 += temp;
				temp33 += temp;
				temp44 += temp;
				temp55 += temp;
				temp66 += temp;
				temp77 += temp;
				temp88 += temp;
				temp99 += temp;
			}
			//copy output to gmem
			int index = blockIdx.x*F*F/2;
			//fill_lower_half_from_registers();
			fill_lower_half_from_registers_float2();
			//symmetric
			if(tile_x!=tile_y){
				//fill_upper_half_from_registers();
				fill_upper_half_from_registers_float2();
			}
		}
	}
}

/*a generic kernel to get the hermitian matrices
 * as the left-hand side of the equations, to update X in ALS
 *examplary F = 100, T = 10
 */
__global__ void
get_hermitianT10(const int batch_offset, float* tt,
		const int* csrRowIndex, const int* csrColIndex, const float lambda, const int m, const int F,
		const float* __restrict__ thetaT) {
	extern __shared__ float2 thetaTemp [];
	int row = blockIdx.x + batch_offset;
	if (row < m) {
		//this block needs to handle end - start thetaT columns
		int start = csrRowIndex[row];
		int end = csrRowIndex[row + 1];
		//slide through [start, end] by window size SCAN_BATCH
		int iterations = (end - start - 1)/SCAN_BATCH + 1;
		float temp0= 0, temp1= 0, temp2= 0, temp3= 0, temp4= 0, temp5= 0, temp6= 0, temp7= 0, temp8= 0, temp9 = 0;
		float temp10= 0, temp11= 0, temp12= 0, temp13= 0, temp14= 0, temp15= 0, temp16= 0, temp17= 0, temp18= 0, temp19 = 0;
		float temp20= 0, temp21= 0, temp22= 0, temp23= 0, temp24= 0, temp25= 0, temp26= 0, temp27= 0, temp28= 0, temp29 = 0;
		float temp30= 0, temp31= 0, temp32= 0, temp33= 0, temp34= 0, temp35= 0, temp36= 0, temp37= 0, temp38= 0, temp39 = 0;
		float temp40= 0, temp41= 0, temp42= 0, temp43= 0, temp44= 0, temp45= 0, temp46= 0, temp47= 0, temp48= 0, temp49 = 0;
		float temp50= 0, temp51= 0, temp52= 0, temp53= 0, temp54= 0, temp55= 0, temp56= 0, temp57= 0, temp58= 0, temp59 = 0;
		float temp60= 0, temp61= 0, temp62= 0, temp63= 0, temp64= 0, temp65= 0, temp66= 0, temp67= 0, temp68= 0, temp69 = 0;
		float temp70= 0, temp71= 0, temp72= 0, temp73= 0, temp74= 0, temp75= 0, temp76= 0, temp77= 0, temp78= 0, temp79 = 0;
		float temp80= 0, temp81= 0, temp82= 0, temp83= 0, temp84= 0, temp85= 0, temp86= 0, temp87= 0, temp88= 0, temp89 = 0;
		float temp90= 0, temp91= 0, temp92= 0, temp93= 0, temp94= 0, temp95= 0, temp96= 0, temp97= 0, temp98= 0, temp99 = 0;

		int N = F/T10; // N = 100/10=10; for F = 100 and T = 10
		int effective_block_size = N*(N+1)/2;
		//get the x and y coordinate
		int tile_x = 0;
		int tile_y = 0;
		for ( int i = 0; i < N; i++ ) {
			int end = ((2*N-i)*(i+1))/2;
			if(threadIdx.x < end){
				tile_x = i * T10;
				tile_y = (N + threadIdx.x - end) * T10;
				break;
			}
		}
		int index = blockIdx.x*F*F;
		//iteration: copy gmem-->smem; aggregate smem-->register
		for (int iter = 0; iter < iterations; iter ++){
			//phase 1 in iteration: gmem --> smem
			
			//REQ: blockDim.x >= F/2
			if(threadIdx.x < F/2){
				for(int k = 0; k< SCAN_BATCH; k++){
					if(iter*SCAN_BATCH + k < end - start){
						float2 theta;
						theta.x = __ldg(&thetaT[F * csrColIndex[start + iter*SCAN_BATCH + k] + 2*threadIdx.x]);
						theta.y = __ldg(&thetaT[F * csrColIndex[start + iter*SCAN_BATCH + k] + 2*threadIdx.x+1]);
						thetaTemp[k * F/2 + threadIdx.x] = theta;
						//this simpler statement is slower.
						//thetaTemp[k * F/2 + threadIdx.x] = __ldg((float2*)&thetaT[F * csrColIndex[start + iter*SCAN_BATCH + k] + 2*threadIdx.x]);
					}
					//not enough theta to copy, set zero
					else
						memset(&thetaTemp[k*F/2 + threadIdx.x], 0, 2*sizeof(float));
				}
			}			
			__syncthreads();
			
			//phase 2 in iteration: smem --> register
			if(threadIdx.x < effective_block_size){//this redundant "if" seems improving kernel performance
				for(int k = 0; k < SCAN_BATCH; k++){
					accumulate_in_registers();
				}
			}
		}
		//end of iteration in copying from smem and aggregating in register
		__syncthreads();

		//phase 3, after iteration: register --> gmem
		if(threadIdx.x < effective_block_size){
			fill_lower_half_from_registers();

			//symmetric
			if(tile_x != tile_y){
				fill_upper_half_from_registers();
			}
			//regularization
			if(tile_x == tile_y){
				for(int k = 0; k < T10; k++)
					tt[index + (tile_x+k)*(1+F)] += (end - start) * lambda;
			}
		}
	}
}


float doALS(const int* csrRowIndexHostPtr, const int* csrColIndexHostPtr, const float* csrValHostPtr,
		const int* cscRowIndexHostPtr, const int* cscColIndexHostPtr, const float* cscValHostPtr,
		const int* cooRowIndexHostPtr, float* thetaTHost, float* XTHost,
		const int * cooRowIndexTestHostPtr, const int * cooColIndexTestHostPtr, const float * cooValHostTestPtr,
		const int m, const int n, const int f, const long nnz, const long nnz_test, const float lambda,
		const int ITERS, const int X_BATCH, const int THETA_BATCH, const int DEVICEID)
{
	hipSetDevice(DEVICEID);
	printf("*******parameters: m: %d, n:  %d, f: %d, nnz: %ld \n", m, n, f, nnz);
	//device pointers
	int * csrRowIndex = 0;
	int * csrColIndex = 0;
	float * csrVal = 0;
	float * thetaT = 0;
	float * tt = 0;
	float * XT = 0;
	float * cscVal =0;
	int * cscRowIndex = 0;
	int * cscColIndex = 0;
	//coo to calculate RMSE
	int * cooRowIndex =0;
	float * cooVal_test;
	int * cooRowIndex_test;
	int * cooColIndex_test;
	float final_rmse = 0;
	printf("*******start allocating memory on GPU...\n");
	cudacall(hipMalloc((void** ) &cscRowIndex,nnz * sizeof(cscRowIndex[0])));
	cudacall(hipMalloc((void** ) &cscColIndex, (n+1) * sizeof(cscColIndex[0])));
	cudacall(hipMalloc((void** ) &cscVal, nnz * sizeof(cscVal[0])));
	//dimension: F*N
	cudacall(hipMalloc((void** ) &thetaT, f * n * sizeof(thetaT[0])));
	//dimension: M*F
	cudacall(hipMalloc((void** ) &XT, f * m * sizeof(XT[0])));

	printf("*******start copying memory to GPU...\n");

	cudacall(hipMemcpy(cscRowIndex, cscRowIndexHostPtr,(size_t ) nnz * sizeof(cscRowIndex[0]), hipMemcpyHostToDevice));
	cudacall(hipMemcpy(cscColIndex, cscColIndexHostPtr,(size_t ) (n+1) * sizeof(cscColIndex[0]), hipMemcpyHostToDevice));
	cudacall(hipMemcpy(cscVal, cscValHostPtr,(size_t ) (nnz * sizeof(cscVal[0])),hipMemcpyHostToDevice));
	cudacall(hipMemcpy(thetaT, thetaTHost, (size_t ) (n * f * sizeof(thetaT[0])), hipMemcpyHostToDevice));
	//CG needs XT
	cudacall(hipMemcpy(XT, XTHost, (size_t ) (m * f * sizeof(XT[0])), hipMemcpyHostToDevice));

	cudacall(hipDeviceSetCacheConfig(hipFuncCachePreferShared));
	//64-bit smem access
	//http://acceleware.com/blog/maximizing-shared-memory-bandwidth-nvidia-kepler-gpus
	hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

	//initialize cublas, cusparse
	hipblasHandle_t handle;
	cublascall(hipblasCreate(&handle));
	hipsparseHandle_t cushandle = 0;
	cusparsecall(hipsparseCreate(&cushandle));
	hipsparseMatDescr_t descr;
	cusparsecall( hipsparseCreateMatDescr(&descr));
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
	using namespace std;
	#ifdef DEBUG
	//variable used to time
	double t0 = 0;
	double t1 = 0;
	#endif

	printf("*******start iterations...\n");
	for(int iter = 0; iter < ITERS ; iter ++){
		#ifdef DEBUG
		printf("---------------------------ALS iteration %d, update X.----------------------------------\n", iter);
		t0 = seconds();
		t1 = seconds();
		#endif
		//copy csr matrix in
		cudacall(hipMalloc((void** ) &csrRowIndex,(m + 1) * sizeof(csrRowIndex[0])));
		cudacall(hipMalloc((void** ) &csrColIndex, nnz * sizeof(csrColIndex[0])));
		cudacall(hipMalloc((void** ) &csrVal, nnz * sizeof(csrVal[0])));
		cudacall(hipMemcpy(csrRowIndex, csrRowIndexHostPtr,(size_t ) ((m + 1) * sizeof(csrRowIndex[0])), hipMemcpyHostToDevice));
		cudacall(hipMemcpy(csrColIndex, csrColIndexHostPtr,(size_t ) (nnz * sizeof(csrColIndex[0])), hipMemcpyHostToDevice));
		cudacall(hipMemcpy(csrVal, csrValHostPtr,(size_t ) (nnz * sizeof(csrVal[0])),hipMemcpyHostToDevice));
		#ifdef DEBUG
		printf("\tgenerate: Y*theta using cusparse.\n");
		#endif
		float * ytheta = 0;
		float * ythetaT = 0;
		cudacall(hipMalloc((void** ) &ytheta, f * m * sizeof(ytheta[0])));
		cudacall(hipMalloc((void** ) &ythetaT, f * m * sizeof(ythetaT[0])));

		const float alpha = 1.0f;
		const float beta = 0.0f;
		cusparsecall (hipsparseScsrmm2(cushandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
				HIPSPARSE_OPERATION_TRANSPOSE, m, f, n, nnz, &alpha, descr, csrVal,
				csrRowIndex, csrColIndex, thetaT, f, &beta, ytheta, m) );
		//hipDeviceSynchronize();
		//printf("*******transpose ytheta use cublas.\n");
		//ytheta: m*f; need ythetaT = (ytheta).T = f*m
		cublascall(hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, f, m, &alpha,
				(const float * ) ytheta, m, &beta, ythetaT, f, ythetaT, f));
		//hipDeviceSynchronize();
		//cudaCheckError();
		cudacall(hipFree(ytheta));
		cudacall(hipFree(csrVal));
		#ifdef DEBUG
		printf("\tgenerate: Y*theta run %f seconds.\n", seconds() - t1);
		#endif

		int block_dim = f/T10*(f/T10+1)/2;
		if (block_dim < f/2) block_dim = f/2;
		for(int batch_id = 0; batch_id< X_BATCH; batch_id ++){
			#ifdef DEBUG
			printf("*******batch %d / %d.*******\n", batch_id, X_BATCH);
			#endif
			int batch_size = 0;
			if(batch_id != X_BATCH - 1)
				batch_size = m/X_BATCH;
			else
				batch_size = m - batch_id*(m/X_BATCH);
			int batch_offset = batch_id * (m/X_BATCH);
			//fprintf(stderr, "Malloc size is - %llu\n", f * f * batch_size * sizeof(float));
			//use fp16 in tt
			//if (tt == NULL)
			#ifdef CUMF_TT_FP16
			cudacall(hipMalloc((void** ) &tt, f/2 * f * batch_size * sizeof(float)));
			#else
			cudacall(hipMalloc((void** ) &tt, f * f * batch_size * sizeof(float)));
			#endif
			#ifdef DEBUG
			t1 = seconds();
			printf("\tupdateXByBlock kernel.\n");
			#endif
			if(f == 100){
				//do not use fp16 by default
				#ifdef CUMF_USE_HALF
				half* thetaT_fp16 = 0;
				cudacall(hipMalloc((void** ) &thetaT_fp16, f * n * sizeof(thetaT_fp16[0])));
				fp32Array2fp16Array<<<(n*f-1)/1024 + 1, 1024>>>(thetaT, thetaT_fp16, f*n);
				get_hermitian100WithHalf<<<batch_size, 64, SCAN_BATCH * f/2*sizeof(float2)>>>
					(batch_offset, tt, csrRowIndex, csrColIndex, lambda, m, f, thetaT_fp16);
				cudacall(hipFree(thetaT_fp16));
				#elif defined(CUMF_TT_FP16)
				get_hermitian100_tt_fp16<<<batch_size, 64, SCAN_BATCH * f/2*sizeof(float2)>>>
					(batch_offset, (half2*) tt, csrRowIndex, csrColIndex, lambda, m, f, (float2*)thetaT);	
					#ifdef CUMF_SAVE_MODEL
					saveDeviceFloatArrayToFile(std::string("./log/cg-xx16-tt16.") + std::to_string(iter),  f * f * batch_size/2, tt);
					#endif					
				#else
				get_hermitian100<<<batch_size, 64, SCAN_BATCH * f/2*sizeof(float2)>>>
					(batch_offset, (float2*)tt, csrRowIndex, csrColIndex, lambda, m, f, (float2*)thetaT);
					#ifdef CUMF_SAVE_MODEL
					saveDeviceFloatArrayToFile(std::string("./log/0904/tt32.") + std::to_string(iter),  f * f * batch_size, tt);
					#endif
				//This commented out is the fused kernel
				//performance not good due to register pressure and low occupancy
				//alsUpdateFeature100Host
				//	(batch_offset, csrRowIndex, csrColIndex, lambda, m, f, thetaT, XT, ythetaT, 6);
				#endif
			}
			else
				get_hermitianT10<<<batch_size, block_dim, SCAN_BATCH * f/2*sizeof(float2)>>>
					(batch_offset, tt, csrRowIndex, csrColIndex, lambda, m, f, thetaT);
			hipDeviceSynchronize();
			cudaCheckError();
			#ifdef DEBUG
			printf("\tupdate X kernel run %f seconds, gridSize: %d, blockSize %d.\n", seconds() - t1, batch_size, f);
			t1 = seconds();
			#endif
			#ifdef USE_CG	//use CG iterative solver
				#ifdef CUMF_TT_FP16
				//cg_iter = als_iter: solve more carefully in later ALS iterations
				printf("\tCG solver with fp16.\n");
				updateXWithCGHost_tt_fp16(tt, &XT[batch_offset*f], &ythetaT[batch_offset*f], batch_size, f, CG_ITER);
				#else
				printf("\tCG solver with fp32.\n");
				updateXWithCGHost(tt, &XT[batch_offset*f], &ythetaT[batch_offset*f], batch_size, f, CG_ITER);
				#endif
			#else//use LU solver instead
			//host pointers for cublas batch operations
			float ** devPtrTTHost = 0;
			cudacall(hipHostMalloc( (void** ) &devPtrTTHost, batch_size * sizeof(*devPtrTTHost) ) );
			float **devPtrYthetaTHost = 0;
			cudacall(hipHostMalloc( (void** ) &devPtrYthetaTHost, batch_size * sizeof(*devPtrYthetaTHost) ) );
			updateX(batch_size, batch_offset, ythetaT, tt, XT, handle, m, n, f, nnz, devPtrTTHost, devPtrYthetaTHost);	
			cudacall(hipHostFree(devPtrTTHost));
			cudacall(hipHostFree(devPtrYthetaTHost));
			#endif
			#ifdef DEBUG
			printf("\tinvoke updateX with batch_size: %d, batch_offset: %d..\n", batch_size, batch_offset);
			printf("\tupdateX solver run seconds: %f \n", seconds() - t1);
			#endif
			cudacall(hipFree(tt));
		}
		#ifdef DEBUG
		printf("update X run %f seconds, gridSize: %d, blockSize %d.\n", seconds() - t0, m, f);
		#endif
		cudacall(hipFree(csrRowIndex));
		cudacall(hipFree(csrColIndex));
		cudacall(hipFree(ythetaT));

///*
		#ifdef DEBUG
		t0 = seconds();
		t1 = seconds();
		printf("---------------------------------- ALS iteration %d, update theta ----------------------------------\n", iter);
		printf("\tgenerate: Y'*X using cusparse.\n");
		#endif
		float * yTX = 0;
		float * yTXT = 0;
		cudacall(hipMalloc((void** ) &yTXT, f * n * sizeof(yTXT[0])));
		cudacall(hipMalloc((void** ) &yTX, n * f * sizeof(yTX[0])));
		cusparsecall( hipsparseScsrmm2(cushandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
				HIPSPARSE_OPERATION_TRANSPOSE, n, f, m, nnz, &alpha, descr, cscVal,
				cscColIndex, cscRowIndex, XT, f, &beta, yTX, n) );
		//hipDeviceSynchronize();
		//printf("*******transpose yTX \n");
		//yTX: n*f; need yTXT = (yTX).T = f*n
		cublascall(hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, f, n, &alpha,
				(const float * ) yTX, n, &beta, yTXT, f, yTXT, f));
		hipDeviceSynchronize();
		cudacall(hipFree(yTX));
		#ifdef DEBUG
		printf("\tgenerate: Y'*X run %f seconds.\n", seconds() - t1);
		#endif
		//in batches, when N is huge
		for(int batch_id = 0; batch_id< THETA_BATCH; batch_id ++){
			#ifdef DEBUG
			printf("*******batch %d / %d.*******\n", batch_id, THETA_BATCH);
			#endif
			int batch_size = 0;
			if(batch_id != THETA_BATCH - 1)
				batch_size = n/THETA_BATCH;
			else
				batch_size = n - batch_id*(n/THETA_BATCH);
			int batch_offset = batch_id * (n/THETA_BATCH);

			float * xx = 0;
			#ifdef CUMF_XX_FP16
			cudacall(hipMalloc((void** ) &xx, f/2 * f * batch_size * sizeof(xx[0])));
			cudacall( hipMemset(xx, 0, f/2*f*batch_size*sizeof(float)) );
			#else
			cudacall(hipMalloc((void** ) &xx, f * f * batch_size * sizeof(xx[0])));
			cudacall( hipMemset(xx, 0, f*f*batch_size*sizeof(float)) );
			#endif
			#ifdef DEBUG
			t1 = seconds();
			printf("\tupdateThetaByBlock kernel.\n");
			#endif
			//get_hermitian_theta<<<batch_size, 64>>>(batch_offset, xx, cscRowIndex, cscColIndex, lambda, n);
			//updateThetaByBlock2pRegDsmemTile<<<batch_size, F>>>
			if(f == 100){
				#ifdef CUMF_USE_HALF
				half * XT_fp16 = 0;
				cudacall(hipMalloc((void** ) &XT_fp16, f * m * sizeof(XT_fp16[0])));
				fp32Array2fp16Array<<<(n*f-1)/1024 + 1, 1024>>>(XT, XT_fp16, f*m);
				get_hermitian100WithHalf<<<batch_size, 64, SCAN_BATCH * f/2*sizeof(float2)>>>
					(batch_offset, xx, cscColIndex, cscRowIndex, lambda, n, f, XT_fp16);
				cudacall(hipFree(XT_fp16));
				#elif defined(CUMF_XX_FP16)
				get_hermitian100_tt_fp16<<<batch_size, 64, SCAN_BATCH * f/2*sizeof(float2)>>>
					(batch_offset, (half2*) xx, cscColIndex, cscRowIndex, lambda, n, f, (float2*)XT);
				#else
				get_hermitian100<<<batch_size, 64, SCAN_BATCH * f/2*sizeof(float2)>>>
					(batch_offset, (float2*)xx, cscColIndex, cscRowIndex, lambda, n, f, (float2*)XT);
				#endif
			}
			else
				get_hermitianT10<<<batch_size, block_dim, SCAN_BATCH*f*sizeof(float)>>>
					(batch_offset, xx, cscColIndex, cscRowIndex, lambda, n, f, XT);
			hipDeviceSynchronize();
			cudaCheckError();
			#ifdef DEBUG
			printf("\tupdate Theta kernel run %f seconds, gridSize: %d, blockSize %d.\n",
					seconds() - t1, batch_size, f);
			t1 = seconds();
			#endif			
			#ifdef DEBUG
			printf("*******invoke updateTheta with batch_size: %d, batch_offset: %d.\n", batch_size, batch_offset);
			#endif
			#ifdef USE_CG
				#ifdef CUMF_XX_FP16
				printf("\tCG solver with fp16.\n");
				updateXWithCGHost_tt_fp16(xx, &thetaT[batch_offset*f], &yTXT[batch_offset*f], batch_size, f, CG_ITER);
				#else
				printf("\tCG solver with fp32.\n");
				updateXWithCGHost(xx, &thetaT[batch_offset*f], &yTXT[batch_offset*f], batch_size, f, CG_ITER);
				#endif
			#else
			float ** devPtrXXHost = 0;
			cudacall(hipHostMalloc( (void** ) &devPtrXXHost, batch_size * sizeof(*devPtrXXHost) ) );
			float **devPtrYTXTHost = 0;
			cudacall(hipHostMalloc( (void** ) &devPtrYTXTHost, batch_size * sizeof(*devPtrYTXTHost) ) );
			updateTheta(batch_size, batch_offset, xx, yTXT, thetaT, handle, m,  n,  f,  nnz,
					devPtrXXHost, devPtrYTXTHost);
			#ifdef CUMF_SAVE_MODEL
			saveDeviceFloatArrayToFile(std::string("./log/0827/lu-xx32.iter") + std::to_string(iter) + std::string(".batch") + std::to_string(batch_id),  f * f * batch_size, xx);
			#endif				
			cudacall(hipHostFree(devPtrXXHost));
			cudacall(hipHostFree(devPtrYTXTHost));
			#endif
			#ifdef DEBUG
			printf("\tupdateTheta solver run seconds: %f \n", seconds() - t1);
			#endif
			cudacall(hipFree(xx));
		}
		cudacall(hipFree(yTXT));
		#ifdef DEBUG
		printf("update theta run %f seconds, gridSize: %d, blockSize %d.\n",
				seconds() - t0, n, f);
		printf("Calculate RMSE.\n");
		#endif
		float * errors_train = 0;
		int error_size = 1000;
		cudacall(hipMalloc((void** ) &errors_train, error_size * sizeof(errors_train[0])));
		cudacall( hipMemset(errors_train, 0, error_size*sizeof(float)) );

		cudacall(hipMalloc((void** ) &cooRowIndex, nnz * sizeof(cooRowIndex[0])));
		cudacall(hipMemcpy(cooRowIndex, cooRowIndexHostPtr,(size_t ) (nnz * sizeof(cooRowIndex[0])), hipMemcpyHostToDevice));
		cudacall(hipMalloc((void** ) &csrColIndex, nnz * sizeof(csrColIndex[0])));
		cudacall(hipMalloc((void** ) &csrVal, nnz * sizeof(csrVal[0])));
		cudacall(hipMemcpy(csrColIndex, csrColIndexHostPtr,(size_t ) (nnz * sizeof(csrColIndex[0])), hipMemcpyHostToDevice));
		cudacall(hipMemcpy(csrVal, csrValHostPtr,(size_t ) (nnz * sizeof(csrVal[0])),hipMemcpyHostToDevice));

		RMSE<<<(nnz-1)/256 + 1, 256>>>
				(csrVal, cooRowIndex, csrColIndex, thetaT, XT, errors_train, nnz, error_size, f);
		hipDeviceSynchronize();
		cudaCheckError();
		cudacall(hipFree(cooRowIndex));
		cudacall(hipFree(csrColIndex));
		cudacall(hipFree(csrVal));

		float* rmse_train = (float*) malloc (sizeof(float));
		cublascall( hipblasSasum(handle, error_size, errors_train, 1, rmse_train) );

		hipDeviceSynchronize();
		printf("--------- Train RMSE in iter %d: %f\n", iter, sqrt((*rmse_train)/nnz));
		cudacall(hipFree(errors_train));

		
		float * errors_test = 0;
		cudacall(hipMalloc((void** ) &errors_test, error_size * sizeof(errors_test[0])));
		cudacall( hipMemset(errors_test, 0, error_size*sizeof(float)) );

		cudacall(hipMalloc((void** ) &cooRowIndex_test, nnz_test * sizeof(cooRowIndex_test[0])));
		cudacall(hipMemcpy(cooRowIndex_test, cooRowIndexTestHostPtr,(size_t ) (nnz_test * sizeof(cooRowIndex_test[0])), hipMemcpyHostToDevice));
		cudacall(hipMalloc((void** ) &cooColIndex_test, nnz_test * sizeof(cooColIndex_test[0])));
		cudacall(hipMalloc((void** ) &cooVal_test, nnz_test * sizeof(cooVal_test[0])));
		cudacall(hipMemcpy(cooColIndex_test, cooColIndexTestHostPtr,(size_t ) (nnz_test * sizeof(cooColIndex_test[0])), hipMemcpyHostToDevice));
		cudacall(hipMemcpy(cooVal_test, cooValHostTestPtr,(size_t ) (nnz_test * sizeof(cooVal_test[0])),hipMemcpyHostToDevice));

		RMSE<<<(nnz_test-1)/256, 256>>>(cooVal_test, cooRowIndex_test, cooColIndex_test, thetaT, XT,
				errors_test, nnz_test, error_size, f);
		hipDeviceSynchronize();
		cudaCheckError();

		cudacall(hipFree(cooRowIndex_test));
		cudacall(hipFree(cooColIndex_test));
		cudacall(hipFree(cooVal_test));

		float* rmse_test = (float*) malloc (sizeof(float));
		cublascall( hipblasSasum(handle, error_size, errors_test, 1, rmse_test) );
		hipDeviceSynchronize();
		final_rmse = sqrt((*rmse_test)/nnz_test);
		printf("--------- Test RMSE in iter %d: %f\n", iter, final_rmse);
		cudacall(hipFree(errors_test));
//*/		
	}
	//copy feature vectors back to host
	cudacall(hipMemcpy(thetaTHost, thetaT, (size_t ) (n * f * sizeof(thetaT[0])), hipMemcpyDeviceToHost));
	cudacall(hipMemcpy(XTHost, XT, (size_t ) (m * f * sizeof(XT[0])), hipMemcpyDeviceToHost));
	cudacall(hipFree(thetaT));
	cudacall(hipFree(XT));
	cudacall(hipFree(cscVal));
	cudacall(hipFree(cscColIndex));
	cudacall(hipFree(cscRowIndex));
	//WARN: do not call hipDeviceReset inside ALS() 
	//because the caller needs to access XT and thetaT which was in the same context
	//cudacall(hipDeviceReset());
	return final_rmse;
}
